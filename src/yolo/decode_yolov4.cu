#include "hip/hip_runtime.h"
/*********************************************************************************
 *      Project: TensorRT-Alpha-ROS                                              *
 *       Author: @FeiYull (https://github.com/FeiYull)                           *
 *     Modified: Xinran Wei (weixr0605@sina.com)                                 *
 *  Modified on: Apr 27, 2023                                                    *
 *                                                                               *
 *  Copyright (c) 2023, Xinran Wei.                                              *
 *  This code file along with the project is published under MIT LICENCE.        *
 *********************************************************************************/


#include "yolo/yolov4.h"

namespace TRTAROS {

__global__ void decode_YOLOV4_device_kernel(int batch_size, int  num_class, int topK, float conf_thresh,
									float* src, int srcWidth, int srcHeight, int srcArea, 
									float* dst, int dstWidth, int dstHeight, int dstArea)
{
	int dx = blockDim.x * blockIdx.x + threadIdx.x; // "srcArea" dim
	int dy = blockDim.y * blockIdx.y + threadIdx.y; // "batch size" dim
	if (dx >= srcHeight || dy >= batch_size)
	{
		return;
	}
	float* pitem = src + dy * srcArea + dx * srcWidth;
	//float objectness = pitem[4]; //  Pr(Object)
	//if (objectness < conf_thresh)
	//{
	//	return;
	//}
	// find max Pr(Classi/Object)
	//float* class_confidence = pitem + 5;  // Pr(Class0/Object)
	float* class_confidence = pitem + 4;    // Pr(Class0/Object)
	float confidence = *class_confidence++; // Pr(Class1/Object)
	int label = 0;
	for (int i = 1; i < num_class; ++i, ++class_confidence)
	{
		if (*class_confidence > confidence)
		{
			confidence = *class_confidence;
			label = i;
		}
	}
	//confidence *= objectness; // Pr(Class0/Object) * Pr(Object)
	if (confidence < conf_thresh)
	{
		return;
	}
	
	// parray:count, box1, box2, box3(count:)
	// parray[0]:count
	// atomicAdd -> count += 1
	// atomicAdd: return old_count
	//int index = atomicAdd(dst + dy * dstArea, 1);
	//assert(dy == 1);
	int index = atomicAdd(dst + dy * dstArea, 1);
	//int index = atomicAdd(&(dst + dy * dstWidth)[0], 1);
	if (index >= topK)
	{
		return;
	}
	//printf("count = %f \n", (dst + dy * dstArea)[0]);
	// xywh -> xyxy
	float cx = *pitem++;
	float cy = *pitem++;
	float width = *pitem++;
	float height = *pitem++;

	/*float left = cx - width * 0.5f;
	float top = cy - height * 0.5f;
	float right = cx + width * 0.5f;
	float bottom = cy + height * 0.5f;*/

	float left = cx;
	float top = cy;
	float right = width;
	float bottom = height;
	// 
	//float* pout_item = dst + dy * dstArea + 1 + index * dstWidth;
	float* pout_item = dst + dy * dstArea + 1 + index * dstWidth;
	*pout_item++ = left; // todo
	*pout_item++ = top;
	*pout_item++ = right;
	*pout_item++ = bottom;

	/**pout_item++ = *pitem++;
	*pout_item++ = *pitem++;
	*pout_item++ = *pitem++;
	*pout_item++ = *pitem++;*/

	*pout_item++ = confidence;
	*pout_item++ = label;
	*pout_item++ = 1;// 1 = keep, 0 = ignore
	//*pout_item = 1;// 1 = keep, 0 = ignore
}

static __device__ float box_iou(
	float aleft, float atop, float aright, float abottom,
	float bleft, float btop, float bright, float bbottom
) {
	float cleft = max(aleft, bleft);
	float ctop = max(atop, btop);
	float cright = min(aright, bright);
	float cbottom = min(abottom, bbottom);

	float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
	if (c_area == 0.0f)
		return 0.0f;

	float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
	float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
	return c_area / (a_area + b_area - c_area);
}

void YOLOV4::decodeDevice(utils::InitParameter param, 
	float* src, int srcWidth, int srcHeight, int srcArea, 
	float* dst, int dstWidth, int dstHeight)
{
	dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid_size((srcHeight + BLOCK_SIZE - 1) / BLOCK_SIZE,
		(param.batch_size + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int dstArea = 1 + dstWidth * dstHeight;
	
	decode_YOLOV4_device_kernel << < grid_size, block_size, 0, nullptr >> >(
		param.batch_size, param.num_class, param.topK, param.conf_thresh,
		src, srcWidth, srcHeight, srcArea, 
		dst, dstWidth, dstHeight, dstArea
	);
}


}