#include "hip/hip_runtime.h"
/*********************************************************************************
 *      Project: TensorRT-Alpha-ROS                                              *
 *       Author: @FeiYull (https://github.com/FeiYull)                           *
 *     Modified: Xinran Wei (weixr0605@sina.com)                                 *
 *  Modified on: Apr 27, 2023                                                    *
 *                                                                               *
 *  Copyright (c) 2023, Xinran Wei.                                              *
 *  This code file along with the project is published under MIT LICENCE.        *
 *********************************************************************************/


#include "yolo/yolov8.h"

namespace TRTAROS {

__global__ void decode_yolov8_device_kernel(int batch_size, int  num_class, int topK, float conf_thresh,
	float* src, int srcWidth, int srcHeight, int srcArea,
	float* dst, int dstWidth, int dstHeight, int dstArea)
{
	int dx = blockDim.x * blockIdx.x + threadIdx.x; // "srcArea" dim
	int dy = blockDim.y * blockIdx.y + threadIdx.y; // "batch size" dim
	if (dx >= srcHeight || dy >= batch_size)
	{
		return;
	}
	float* pitem = src + dy * srcArea + dx * srcWidth;

	// find max Pr(Classi/Object)
	//float* class_confidence = pitem + 5;  // Pr(Class0/Object)
	float* class_confidence = pitem + 4;    // Pr(Class0/Object)
	float confidence = *class_confidence++; // Pr(Class1/Object)
	int label = 0;
	for (int i = 1; i < num_class; ++i, ++class_confidence)
	{
		if (*class_confidence > confidence)
		{
			confidence = *class_confidence;
			label = i;
		}
	}
	if (confidence < conf_thresh)
	{
		return;
	}

	// parray:count, box1, box2, box3(count:)
	// parray[0]:count
	// atomicAdd -> count += 1
	// atomicAdd: return old_count
	//int index = atomicAdd(dst + dy * dstArea, 1);
	//assert(dy == 1);
	int index = atomicAdd(dst + dy * dstArea, 1);

	if (index >= topK)
	{
		return;
	}
	// xywh -> xyxy
	float cx = *pitem++;
	float cy = *pitem++;
	float width = *pitem++;
	float height = *pitem++;

	float left = cx - width * 0.5f;
	float top = cy - height * 0.5f;
	float right = cx + width * 0.5f;
	float bottom = cy + height * 0.5f;

	/*float left = cx;
	float top = cy;
	float right = width;
	float bottom = height;*/
	float* pout_item = dst + dy * dstArea + 1 + index * dstWidth;
	*pout_item++ = left; // todo
	*pout_item++ = top;
	*pout_item++ = right;
	*pout_item++ = bottom;


	*pout_item++ = confidence;
	*pout_item++ = label;
	*pout_item++ = 1;// 1 = keep, 0 = ignore
}

void YOLOV8::decodeDevice(utils::InitParameter param, float* src, int srcWidth, 
	int srcHeight, int srcArea, float* dst, int dstWidth, int dstHeight)
{
	dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid_size((srcHeight + BLOCK_SIZE - 1) / BLOCK_SIZE,
		(param.batch_size + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int dstArea = 1 + dstWidth * dstHeight;

	decode_yolov8_device_kernel << < grid_size, block_size, 0, nullptr >> > (param.batch_size, param.num_class, param.topK, param.conf_thresh,
		src, srcWidth, srcHeight, srcArea,
		dst, dstWidth, dstHeight, dstArea);
}


__global__ void transpose_device_kernel(int batch_size,
	float* src, int srcWidth, int srcHeight, int srcArea,
	float* dst, int dstWidth, int dstHeight, int dstArea)
{
	int dx = blockDim.x * blockIdx.x + threadIdx.x; // "srcArea" dim
	int dy = blockDim.y * blockIdx.y + threadIdx.y; // "batch size" dim
	if (dx >= dstHeight || dy >= batch_size)
	{
		return;
	}
	float* p_dst_row = dst + dy * dstArea + dx * dstWidth; // row = dx
	float* p_src_col = src + dy * srcArea + dx; // col = dx

	for (int i = 0; i < dstWidth; i++)
	{
		p_dst_row[i] = p_src_col[i * srcWidth];
	}
}

/*
	src:
				8400 ->
	84	  x1 x2  ...... x8400
	 |    y1 y2  ...... y8400
	 V	  w1 w2  ...... w8400
		  h1 h2  ...... h8400
		  c0  .
		  c1  .
		  c2  .
		  .   .
		  .   .
		  .
		  c79

*/
void YOLOV8::transposeDevice(utils::InitParameter param, 
	float* src, int srcWidth, int srcHeight, int srcArea, 
	float* dst, int dstWidth, int dstHeight)
{
	dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid_size((dstHeight + BLOCK_SIZE - 1) / BLOCK_SIZE,
		(param.batch_size + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int dstArea = dstWidth * dstHeight;

	transpose_device_kernel << < grid_size, block_size, 0, nullptr >> > (param.batch_size,
		src, srcWidth, srcHeight, srcArea,
		dst, dstWidth, dstHeight, dstArea);
}

}